﻿#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t mergeSortWithCuda(int* arr, unsigned int size);

__global__ void mergeSortKernel(int *arr, int *aux, unsigned int blockSize, const unsigned int last)
{
    int x = threadIdx.x;
    int start = blockSize * x;
    int end = start + blockSize - 1;
    int mid = start + (blockSize / 2) - 1;
    int l = start, r = mid + 1, i = start;

    if (end > last) { end = last; }
    if (start == end || end <= mid) { return; }

    while (l <= mid && r <= end) {
        if (arr[l] <= arr[r]) {
            aux[i++] = arr[l++];
        }
        else {
            aux[i++] = arr[r++];
        }
    }

    while (l <= mid) { aux[i++] = arr[l++]; }
    while (r <= end) { aux[i++] = arr[r++]; }

    for (i = start; i <= end; i++) {
        arr[i] = aux[i];
    }
}

hipError_t mergeSortWithCuda(int *arr, unsigned int size)
{
    int *dev_arr = 0;
    int *dev_aux = 0;
    const unsigned int last = size - 1;
    unsigned int threadCount;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for two vectors (main and aux array).
    cudaStatus = hipMalloc((void**)&dev_arr, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_aux, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_arr, arr, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    for (unsigned int blockSize = 2; blockSize < 2 * size; blockSize *= 2) {
        threadCount = size / blockSize;
        if (size % blockSize > 0) { threadCount++; }

        // Launch a kernel on the GPU with one thread for each block.
        mergeSortKernel<<<1, threadCount>>>(dev_arr, dev_aux, blockSize, last);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "mergeSortKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // cudaDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching mergeSortKernel!\n", cudaStatus);
            goto Error;
        }
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(arr, dev_arr, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_arr);
    hipFree(dev_aux);

    return cudaStatus;
}

