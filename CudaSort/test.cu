﻿#include "merge_sort.cuh"

#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define OK 1
#define EXPECTATION_ERROR 1
#define MALLOC_ERROR 2
#define CUDA_ERROR 3

bool assertArrEq(int* expected, int* actual, size_t size);
int testMergeSortWithCuda(int* actual, int* expected, const unsigned int count);

int main()
{
    const unsigned int count = rand() % 2048;
    const unsigned int size = count * sizeof(int);
    int status = MALLOC_ERROR;
    int* actual = (int*) malloc(size);
    int* expected = (int*) malloc(size);

    if (actual != NULL && expected != NULL) {
        status = testMergeSortWithCuda(actual, expected, count);
    }
    else {
        fprintf(stderr, "malloc failed!");
    }

    free(actual);
    free(expected);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    int cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return CUDA_ERROR;
    }

    return status;
}

int cmpInt(const void* a, const void* b) {
    return *(int*)a - *(int*)b;
}

int testMergeSortWithCuda(int* actual, int* expected, const unsigned int count) {
    for (unsigned int i = 0; i < count; i++) {
        expected[i] = actual[i] = rand();
    }

    qsort(expected, count, sizeof(int), cmpInt);

    hipError_t cudaStatus = mergeSortWithCuda(actual, count);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "mergeSortWithCuda failed!");
        return CUDA_ERROR;
    }

    if (!assertArrEq(expected, actual, count * sizeof(int))) {
        puts("cuda sorted array is not equal to the qsorted array!");
        return EXPECTATION_ERROR;
    }

    puts("test ok.");
    return OK;
}

bool assertArrEq(int* expected, int* actual, size_t size) {
    if (memcmp(expected, actual, size) != 0) {
        return false;
    }

    return true;
}
